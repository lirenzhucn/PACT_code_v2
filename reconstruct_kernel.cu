
#include <hip/hip_runtime.h>
#include <math.h>

#define SIGN(x) ((x) > 0.0 ? 1 : -1)

__global__ void init_image_kernel(float *img) {
  size_t xi = blockIdx.x;
  size_t yi = blockIdx.y;
  size_t zi = threadIdx.x;
  size_t imgIdx = zi + yi*blockDim.x + xi*blockDim.x*gridDim.y;
  img[imgIdx] = 0.0;
}

__global__ void calculate_cos_alpha_and_tempc
(float *cosAlpha, float *tempc, float *xRange, float *yRange,
 float *xReceive, float *yReceive, float lenR) {
  size_t xi = blockIdx.x;
  size_t yi = blockIdx.y;
  size_t ni = threadIdx.x;
  size_t idx = ni + yi*blockDim.x + xi*blockDim.x*gridDim.y;
  float dx = xRange[xi] - xReceive[ni];
  float dy = yRange[yi] - yReceive[ni];
  float r0 = sqrt(xReceive[ni]*xReceive[ni] + yReceive[ni]*yReceive[ni]);
  float rr0 = sqrt(dx*dx + dy*dy);
  cosAlpha[idx] = fabs((-xReceive[ni]*dx-yReceive[ni]*dy)/r0/rr0);
  tempc[idx] = rr0 - lenR/cosAlpha[idx];
}

__global__ void backprojection_kernel_fast
(float *img, float *paDataLine,
 float *cosAlpha_, float *tempc_, float *zRange,
 float zReceive, float lenR, float elementHeight,
 float vm, float delayIdx, float fs,
 unsigned int ni, unsigned int nSteps, unsigned int lineLength) {
  size_t xi = blockIdx.x;
  size_t yi = blockIdx.y;
  size_t zi = threadIdx.x;
  size_t precompIdx = ni + yi*nSteps + xi*nSteps*gridDim.y;
  float dz = zRange[zi] - zReceive;
  float cosAlpha = cosAlpha_[precompIdx];
  float tempc = tempc_[precompIdx];
  if (fabs(dz/tempc) < fabs(elementHeight*cosAlpha/2.0/lenR)) {
    size_t imgIdx = zi + yi*blockDim.x + xi*blockDim.x*gridDim.y;
    float rr0 = sqrt(tempc*tempc + dz*dz)*SIGN(tempc) + lenR/cosAlpha;
    float angleWeightB = tempc/sqrt(tempc*tempc+dz*dz)*cosAlpha/(rr0*rr0);
    size_t idx0 = lround((rr0/vm-delayIdx)*fs);
    if (idx0 < lineLength) {
      img[imgIdx] += paDataLine[idx0] / angleWeightB;
    }
  }
}

__global__ void backprojection_2d_kernel_fast
(float *pa_img, float *pa_data, unsigned int *idxAll, float *angularWeight,
 int nSteps, int nTimeSamples) {
  size_t xi = blockIdx.x;
  size_t yi = blockIdx.y;
  size_t zi = threadIdx.x;
  size_t imgIdx = xi + yi*gridDim.x + zi*gridDim.x*gridDim.y;
  // all data arrays are in 'F' order
  for (size_t iStep = 0; iStep < nSteps; iStep++) {
    size_t idx = xi + yi*gridDim.x + iStep*gridDim.x*gridDim.y;
    pa_img[imgIdx] +=
      pa_data[(size_t)(idxAll[idx] - 1 + nTimeSamples*iStep + zi*nTimeSamples*nSteps)] * angularWeight[idx];
  }
}
